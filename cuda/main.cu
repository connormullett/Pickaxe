#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>

__global__ void sha256_cuda(BYTE *data, BYTE *digest, int n)
{
	// perform sha256 calculation here
	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, data, n);
	sha256_final(&ctx, digest);
}

// void sha_256(BYTE *data, BYTE *digest, int n)
// {
// 	compy symbols
// 	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
// 	sha256_cuda(data, digest, n);
// }
